#include "hip/hip_runtime.h"
#include "matrix.h"
#include "polynomial.h"
#include <hip/hip_runtime.h>
#include <random>
#include <sstream>
#include <cstdio>

#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// CUDA核函数 - 矩阵加法
__global__ void matrixAddKernel(const Matrix* A, const Matrix* B, Matrix* C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < A->getCols() && idy < A->getRows()) {
        C->at(idy, idx) = A->at(idy, idx) + B->at(idy, idx);
    }
}

// // CUDA核函数 - 矩阵乘法
// __global__ void matrixMultiplyKernel(const Matrix* A, const Matrix* B, Matrix* C) {
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if (row < A->getRows() && col < B->getCols()) {
//         FiniteField sum = FiniteField::fromParts(0,0);
        
//         for (int k = 0; k < A->getCols(); k++) {
//             FiniteField a = A->at(row, k);
//             FiniteField b = B->at(k, col);
//             FiniteField prod = a * b;
//             sum = sum + prod;
//         }
        
//         C->at(row, col) = sum;
//     }
// }

// 构造函数
__host__ __device__ Matrix::Matrix(int r, int c) : rows(r), cols(c), is_device(false) {
    #ifdef __CUDA_ARCH__
        // 在设备端
        data = (FiniteField*)malloc(rows * cols * sizeof(FiniteField));
        if (data == nullptr) {
            printf("Device memory allocation failed\n");  // 添加调试输出
            rows = 0;
            cols = 0;
            return;
        }
        // 添加内存初始化验证
        for (int i = 0; i < rows * cols; i++) {
            data[i] = FiniteField();
            // 验证初始化是否成功
            if (i == 0) {
                FiniteField test = data[i];
                // 验证加法操作
                FiniteField sum = test + test;
            }
        }
    #else
        // 在主机端
        hipError_t err = hipHostMalloc(&data, rows * cols * sizeof(FiniteField));
        if (err != hipSuccess || data == nullptr) {
            printf("Host memory allocation failed: %s\n", 
                   hipGetErrorString(err));  // 添加错误信息
            throw std::runtime_error("Failed to allocate host memory for matrix");
        }
        // 添加内存初始化验证
        for (int i = 0; i < rows * cols; i++) {
            new (&data[i]) FiniteField();
            // 验证初始化是否成功
            if (i == 0) {
                FiniteField test = data[i];
                // 验证加法操作
                FiniteField sum = test + test;
            }
        }
    #endif
}

// 拷贝构造函数
__host__ __device__ Matrix::Matrix(const Matrix& other) : 
    rows(other.rows), cols(other.cols), is_device(other.is_device) {
    size_t size = rows * cols * sizeof(FiniteField);
    
    #ifdef __CUDA_ARCH__
        data = (FiniteField*)malloc(size);
        if (data != nullptr) {
            memcpy(data, other.data, size);
        }
    #else
        if (is_device) {
            hipMalloc(&data, size);
        } else {
            hipHostMalloc(&data, size);
        }
        if (data != nullptr) {
            if (is_device) {
                hipMemcpy(data, other.data, size, hipMemcpyDeviceToDevice);
            } else {
                memcpy(data, other.data, size);
            }
        }
    #endif
}

// 赋值运算符
__host__ __device__ Matrix& Matrix::operator=(const Matrix& other) {
    if (this != &other) {
        // 先释放原有内存
        if (data != nullptr) {
            #ifdef __CUDA_ARCH__
                free(data);
            #else
                if (is_device) {
                    hipFree(data);
                } else {
                    hipHostFree(data);
                }
            #endif
        }
        
        rows = other.rows;
        cols = other.cols;
        is_device = other.is_device;
        
        size_t size = rows * cols * sizeof(FiniteField);
        
        #ifdef __CUDA_ARCH__
            data = (FiniteField*)malloc(size);
            if (data != nullptr) {
                memcpy(data, other.data, size);
            }
        #else
            if (is_device) {
                hipMalloc(&data, size);
            } else {
                hipHostMalloc(&data, size);
            }
            if (data != nullptr) {
                if (is_device) {
                    hipMemcpy(data, other.data, size, hipMemcpyDeviceToDevice);
                } else {
                    memcpy(data, other.data, size);
                }
            }
        #endif
    }
    return *this;
}

// 在设备上创建矩阵
__host__ Matrix Matrix::createDeviceMatrix(int rows, int cols) {
    Matrix matrix(rows, cols);
    CHECK_CUDA_ERROR(hipMalloc(&matrix.data, rows * cols * sizeof(FiniteField)));
    matrix.is_device = true;
    return matrix;
}

// GPU矩阵加法
__host__ Matrix Matrix::deviceAdd(const Matrix& A, const Matrix& B) {
    if (A.cols != B.cols || A.rows != B.rows) {
        throw std::runtime_error("Matrix dimensions do not match");
    }
    
    if (!A.is_device || !B.is_device) {
        throw std::runtime_error("Input matrices must be on device");
    }
    
    Matrix C = createDeviceMatrix(A.rows, A.cols);
    
    Matrix *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc(&d_A, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, sizeof(Matrix)));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_A, &A, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, &B, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_C, &C, sizeof(Matrix), hipMemcpyHostToDevice));
    
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (A.getCols() + blockSize.x - 1) / blockSize.x,
        (A.getRows() + blockSize.y - 1) / blockSize.y
    );
    
    matrixAddKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return C;
}

// GPU矩阵乘法
__host__ Matrix Matrix::deviceMultiply(const Matrix& A, const Matrix& B) {
    if (A.cols != B.rows) {
        throw std::runtime_error("Matrix dimensions do not match for multiplication");
    }
    
    if (!A.is_device || !B.is_device) {
        throw std::runtime_error("Input matrices must be on device");
    }
    
    Matrix C = createDeviceMatrix(A.rows, B.cols);
    
    // 只需要复制矩阵对象，因为数据已经在设备上
    Matrix *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc(&d_A, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, sizeof(Matrix)));
    
    // 创建临时Matrix对象，确保data指针指向设备内存
    Matrix h_A = A;
    Matrix h_B = B;
    Matrix h_C = C;
    
    CHECK_CUDA_ERROR(hipMemcpy(d_A, &h_A, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, &h_B, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_C, &h_C, sizeof(Matrix), hipMemcpyHostToDevice));
    
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (B.cols + blockSize.x - 1) / blockSize.x,
        (A.rows + blockSize.y - 1) / blockSize.y
    );
    
    // 添加调试信息
    printf("Host: Matrix multiply - A rows=%d, cols=%d, B rows=%d, cols=%d\n",
           A.rows, A.cols, B.rows, B.cols);
           
    size_t sharedMemSize = 2 * blockSize.x * blockSize.y * sizeof(FiniteField);
    matrixMultiplyKernel<<<gridSize, blockSize, sharedMemSize>>>(d_A, d_B, d_C);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return C;
}

// 内存拷贝方法
__host__ void Matrix::copyToDevice(Matrix& d_matrix) const {
    size_t size = rows * cols * sizeof(FiniteField);
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.data, data, size, hipMemcpyHostToDevice));
}

__host__ void Matrix::copyFromDevice(const Matrix& d_matrix) {
    if (!d_matrix.is_device) {
        throw std::runtime_error("Source matrix is not a device matrix");
    }
    
    if (d_matrix.rows != rows || d_matrix.cols != cols) {
        throw std::runtime_error("Matrix dimensions do not match for copy");
    }
    
    size_t size = rows * cols * sizeof(FiniteField);
    CHECK_CUDA_ERROR(hipMemcpy(data, d_matrix.data, size, hipMemcpyDeviceToHost));
}

// 随机化矩阵元素
__host__ void Matrix::randomize() {
    std::random_device rd;
    std::mt19937_64 gen(rd());
    std::uniform_int_distribution<uint64_t> dis;
    
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            uint128_t random_value = dis(gen);
            random_value = (random_value << 64) | dis(gen);
            at(i, j) = FiniteField::fromParts(0, random_value);
        }
    }
}

// 转换为字符串
__host__ std::string Matrix::toString() const {
    std::ostringstream oss;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            oss << "M[" << i << "][" << j << "] = ";
            data[i * cols + j].print();
            oss << "\n";
        }
    }
    return oss.str();
}

// 添加一个辅助函数来验证设备内存
__host__ bool Matrix::validateDevicePointer() const {
    hipPointerAttribute_t attrs;
    return hipPointerGetAttributes(&attrs, data) == hipSuccess && 
           attrs.type == hipMemoryTypeDevice;
}

__host__ __device__ bool Matrix::isInvertible() const {
    if (rows != cols) {
        return false;  // 非方阵一定不可逆
    }
    
    // 已经解决列数大于8问题
    // #ifdef __CUDA_ARCH__
    // if (rows > 8) {
    //     // 在设备端，我们不能抛出异常
    //     return false;
    // }
    // #else
    // if (rows > 4) {
    //     throw std::runtime_error("Matrix size larger than 4x4 is not supported");
    // }
    // #endif
    
    FiniteField det = determinant();
    return !(det == FiniteField::fromParts(0,0));
}

__device__ __host__ FiniteField Matrix::determinant() const {
    if (rows != cols) {
        return FiniteField::fromParts(0,0);
    }
    
    // 1x1到4x4矩阵原直接计算方式
    if (rows <= 4) {
        return determinantSmall();
    }
    
    // 对于更大的矩阵，使用LU分解
    return determinantLU();
}

__device__ __host__ FiniteField Matrix::determinantSmall() const {
    if (rows == 1) return at(0, 0);
    if (rows == 2) return at(0,0) * at(1,1) - at(0,1) * at(1,0);
    
    if (rows == 3) {
        FiniteField det;
        // 正项
        det = det + (at(0,0) * at(1,1) * at(2,2));
        det = det + (at(0,1) * at(1,2) * at(2,0));
        det = det + (at(0,2) * at(1,0) * at(2,1));
        // 负项
        det = det - (at(0,2) * at(1,1) * at(2,0));
        det = det - (at(0,1) * at(1,0) * at(2,2));
        det = det - (at(0,0) * at(1,2) * at(2,1));
        return det;
    }
    
    if (rows == 4) {
        FiniteField det;
        for (int j = 0; j < 4; j++) {
            FiniteField minor_det;
            if (j == 0) {
                minor_det = (at(1,1) * at(2,2) * at(3,3) + 
                           at(1,2) * at(2,3) * at(3,1) + 
                           at(1,3) * at(2,1) * at(3,2)) -
                          (at(1,3) * at(2,2) * at(3,1) + 
                           at(1,2) * at(2,1) * at(3,3) + 
                           at(1,1) * at(2,3) * at(3,2));
            } else if (j == 1) {
                minor_det = (at(1,0) * at(2,2) * at(3,3) + 
                           at(1,2) * at(2,3) * at(3,0) + 
                           at(1,3) * at(2,0) * at(3,2)) -
                          (at(1,3) * at(2,2) * at(3,0) + 
                           at(1,2) * at(2,0) * at(3,3) + 
                           at(1,0) * at(2,3) * at(3,2));
                minor_det = -minor_det;
            } else if (j == 2) {
                minor_det = (at(1,0) * at(2,1) * at(3,3) + 
                           at(1,1) * at(2,3) * at(3,0) + 
                           at(1,3) * at(2,0) * at(3,1)) -
                          (at(1,3) * at(2,1) * at(3,0) + 
                           at(1,1) * at(2,0) * at(3,3) + 
                           at(1,0) * at(2,3) * at(3,1));
            } else {
                minor_det = (at(1,0) * at(2,1) * at(3,2) + 
                           at(1,1) * at(2,2) * at(3,0) + 
                           at(1,2) * at(2,0) * at(3,1)) -
                          (at(1,2) * at(2,1) * at(3,0) + 
                           at(1,1) * at(2,0) * at(3,2) + 
                           at(1,0) * at(2,2) * at(3,1));
                minor_det = -minor_det;
            }
            det = det + (at(0,j) * minor_det);
        }
        return det;
    }
    
    return FiniteField::fromParts(0,0);
}

__device__ __host__ FiniteField Matrix::determinantLU() const {
    // 使用栈内存代替设备内存
    FiniteField L_data[64];  // 支持最大8x8矩阵
    FiniteField U_data[64];
    
    // 初化L和U
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            if(i == j) {
                L_data[idx] = FiniteField::fromParts(0,1);
            } else {
                L_data[idx] = FiniteField::fromParts(0,0);
            }
            U_data[idx] = FiniteField::fromParts(0,0);
        }
    }
    
    // LU分解
    for(int i = 0; i < rows; i++) {
        // 计算U的第i行
        for(int j = i; j < cols; j++) {
            FiniteField sum = FiniteField::fromParts(0,0);
            for(int k = 0; k < i; k++) {
                sum = sum + L_data[i * cols + k] * U_data[k * cols + j];
            }
            U_data[i * cols + j] = at(i,j) - sum;
        }
        
        // 检查对线素是否为0
        if(U_data[i * cols + i].isZero()) {
            return FiniteField::fromParts(0,0); // 矩阵奇
        }
        
        // 计算L的第i列
        FiniteField u_ii_inv = U_data[i * cols + i].inverse();
        for(int j = i + 1; j < rows; j++) {
            FiniteField sum = FiniteField::fromParts(0,0);
            for(int k = 0; k < i; k++) {
                sum = sum + L_data[j * cols + k] * U_data[k * cols + i];
            }
            L_data[j * cols + i] = (at(j,i) - sum) * u_ii_inv;
        }
    }
    
    // 计算行列式：对角线元素的乘
    FiniteField det = FiniteField::fromParts(0,1);
    for(int i = 0; i < rows; i++) {
        det = det * U_data[i * cols + i];
    }
    
    return det;
}

__host__ __device__ Matrix::~Matrix() {
    if (data != nullptr) {
        #ifdef __CUDA_ARCH__
            // 在设备端
            free(data);
        #else
            // 在主机端
            if (is_device) {
                hipFree(data);
            } else {
                hipHostFree(data);  // 使用hipHostFree替代delete[]
            }
        #endif
        data = nullptr;
    }
}

__device__ __host__ Polynomial Matrix::characteristicPolynomial() const {
    #ifdef __CUDA_ARCH__
        return characteristicPolynomialDevice();
    #else 
        printf("Host: Starting characteristic polynomial calculation\n");
        const int n = rows;
        Polynomial charPoly;
        
        FiniteField c[MAX_MATRIX_SIZE + 1];
        FiniteField p[MAX_MATRIX_SIZE + 1];
        
        for (int i = 0; i <= n; ++i) {
            c[i] = FiniteField::fromParts(0, 0);
            p[i] = FiniteField::fromParts(0, 0);
        }
        
        c[n] = FiniteField::fromParts(0, 1);
        
        for (int k = 1; k <= n; ++k) {
            printf("Host: Computing power %d\n", k);
            Matrix Ak = this->power(k);
            printf("Host: Matrix power %d computed, calculating trace\n", k);
            p[k] = Ak.trace().mod();
            printf("Host: Trace for power %d = ", k);
            p[k].print();
            printf("\n");
            
            FiniteField sum = FiniteField::fromParts(0, 0);
            for (int j = 1; j < k; ++j) {
                sum = (sum + c[n - j] * p[k - j]).mod();
            }
            c[n - k] = ((-p[k] - sum) / FiniteField::fromParts(0, k)).mod();
            printf("Host: Coefficient c[%d] = ", n-k);
            c[n - k].print();
            printf("\n");
        }
        
        for (int i = 0; i <= n; ++i) {
            charPoly.setCoefficient(i, c[i]);
        }
        
        return charPoly;
    #endif
}




__device__ __host__ Polynomial Matrix::minimalPolynomial() const {
    // 首先计算特征多项式
    Polynomial charPoly = characteristicPolynomial();
    
    // 计算导数
    Polynomial derivative;
    for(int i = 1; i <= charPoly.degree(); i++) {
        if(i % 2 == 1) {  // 在特征为2的域中，只有奇次项的导数非零
            derivative.setCoefficient(i-1, charPoly[i]);
        }
    }
    
    // 计算GCD并迭代消除重复因子
    Polynomial minPoly = charPoly;
    Polynomial gcd = minPoly.gcd(derivative);
    
    // 检查 gcd 是否为常数多项式（degree = 0）且系数为1
    while(gcd.degree() > 0 || (gcd.degree() == 0 && !(gcd[0] == FiniteField::fromParts(0, 1)))) {
        minPoly = minPoly / gcd;
        derivative = Polynomial();
        for(int i = 1; i <= minPoly.degree(); i++) {
            if(i % 2 == 1) {
                derivative.setCoefficient(i-1, minPoly[i]);
            }
        }
        gcd = minPoly.gcd(derivative);
    }
    
    minPoly.normalize();
    return minPoly;
}

__device__ Polynomial Matrix::characteristicPolynomialDevice() const {
    printf("Device: Starting characteristic polynomial calculation\n");
    const int n = rows;
    Polynomial charPoly;
    
    FiniteField c[MAX_MATRIX_SIZE + 1];
    FiniteField p[MAX_MATRIX_SIZE + 1];
    
    for (int i = 0; i <= n; ++i) {
        c[i] = FiniteField::fromParts(0, 0);
        p[i] = FiniteField::fromParts(0, 0);
    }
    
    c[n] = FiniteField::fromParts(0, 1);
    
    for (int k = 1; k <= n; ++k) {
        printf("Device: Computing power %d\n", k);
        Matrix Ak = this->devicePower(k);
        printf("Device: Matrix power %d computed, calculating trace\n", k);
        p[k] = Ak.deviceTrace();
        printf("Device: Trace for power %d = ", k);
        p[k].print();
        printf("\n");
        
        FiniteField sum = FiniteField::fromParts(0, 0);
        for (int j = 1; j < k; ++j) {
            sum = (sum + c[n - j] * p[k - j]).mod();
        }
        c[n - k] = ((-p[k] - sum) / FiniteField::fromParts(0, k)).mod();
        printf("Device: Coefficient c[%d] = ", n-k);
        c[n - k].print();
        printf("\n");
    }
    
    for (int i = 0; i <= n; ++i) {
        charPoly.setCoefficient(i, c[i]);
    }
    
    return charPoly;
}



__device__ __host__ Matrix Matrix::operator*(const Matrix& other) const {
    if (cols != other.rows) {
        #ifdef __CUDA_ARCH__
        return Matrix();  // 在设备端返回空矩阵
        #else
        throw std::runtime_error("Matrix dimensions mismatch for multiplication");
        #endif
    }

    Matrix result(rows, other.cols);
    
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < other.cols; j++) {
            FiniteField sum = FiniteField::fromParts(0, 0);
            for (int k = 0; k < cols; k++) {
                sum = sum + (at(i, k) * other.at(k, j));
            }
            result.at(i, j) = sum;
        }
    }
    
    return result;
}

__device__ __host__ Matrix Matrix::power(int k) const {
    if (rows != cols) {
        #ifdef __CUDA_ARCH__
        return Matrix();
        #else
        throw std::runtime_error("Matrix must be square to compute power");
        #endif
    }

    // 初始化单位矩阵
    Matrix result(rows, cols);
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            result.at(i, j) = (i == j) ? FiniteField::fromParts(0, 1) : FiniteField::fromParts(0, 0);
        }
    }

    // 拷贝当前矩阵
    Matrix base = *this;

    // 快速幂算法
    while (k > 0) {
        if (k % 2 == 1) {
            result = result * base; // 奇数时将 base 累乘到结果
        }
        base = base * base; // base 平方
        k /= 2; // 指数减半
    }

    return result;
}



__device__ __host__ FiniteField Matrix::trace() const {
    if (rows != cols) {
        #ifdef __CUDA_ARCH__
        return FiniteField::fromParts(0, 0);
        #else
        throw std::runtime_error("Matrix must be square to compute trace");
        #endif
    }

    FiniteField trace = FiniteField::fromParts(0, 0);

    // 累加主对角线元素
    for (int i = 0; i < rows; ++i) {
        trace = (trace + at(i, i)).mod(); // 每次累加后取模
    }

    return trace;
}

__global__ void matrixMultiplyKernel(const Matrix* A, const Matrix* B, Matrix* C) {
    extern __shared__ FiniteField sharedMem[];
    FiniteField* As = sharedMem;
    FiniteField* Bs = &sharedMem[blockDim.x * blockDim.y];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // 初始化累加和
    FiniteField sum = FiniteField::fromParts(0, 0);
    
    // 用于调试
    if (row == 0 && col == 0) {
        printf("Debug: Matrix multiply - A rows=%d, cols=%d, B rows=%d, cols=%d\n",
               A->getRows(), A->getCols(), B->getRows(), B->getCols());
    }
    
    for (int m = 0; m < (A->getCols() + blockDim.x - 1) / blockDim.x; ++m) {
        // 加载数据到共享内存
        if (row < A->getRows() && m * blockDim.x + threadIdx.x < A->getCols()) {
            As[threadIdx.y * blockDim.x + threadIdx.x] = A->at(row, m * blockDim.x + threadIdx.x);
        }
        
        if (m * blockDim.y + threadIdx.y < B->getRows() && col < B->getCols()) {
            Bs[threadIdx.y * blockDim.x + threadIdx.x] = B->at(m * blockDim.y + threadIdx.y, col);
        }
        
        __syncthreads();
        
        // 计算当前块的部分和
        if (row < C->getRows() && col < C->getCols()) {
            for (int k = 0; k < blockDim.x && m * blockDim.x + k < A->getCols(); ++k) {
                sum = (sum + As[threadIdx.y * blockDim.x + k] * 
                      Bs[k * blockDim.x + threadIdx.x]).mod();
            }
        }
        
        __syncthreads();
    }
    
    // 写入结果
    if (row < C->getRows() && col < C->getCols()) {
        C->at(row, col) = sum.mod();
        // 调试输出
        if (row == 0 && col == 0) {
            printf("Debug: First element of result = ");
            sum.print();
            printf("\n");
        }
    }
}

__device__ Matrix Matrix::devicePower(int k) const {
    printf("Device: Starting power calculation for k=%d\n", k);
    if (k == 0) return Matrix::identity(this->getRows());
    if (k == 1) return *this;
    
    // 打印当前矩阵的维度信息
    printf("Device: Current matrix dimensions: rows=%d, cols=%d\n", rows, cols);
    
    // 直接在设备栈上创建矩阵
    Matrix result = Matrix::identity(this->getRows());
    Matrix base = *this;
    Matrix temp(rows, cols);  // 临时矩阵用于存储中间结果
    
    while (k > 0) {
        if (k & 1) {
            printf("Device: Multiplying in power calculation, k=%d\n", k);
            
            // 打印矩阵维度信息
            printf("Device: Matrix multiply dimensions - result(%d,%d) * base(%d,%d)\n",
                   result.rows, result.cols, base.rows, base.cols);
                   
            dim3 blockSize(16, 16);
            dim3 gridSize(
                (cols + blockSize.x - 1) / blockSize.x,
                (rows + blockSize.y - 1) / blockSize.y
            );
            
            size_t sharedMemSize = 2 * blockSize.x * blockSize.y * sizeof(FiniteField);
            
            // 直接计算矩阵乘法，不使用kernel
            for (int i = 0; i < rows; i++) {
                for (int j = 0; j < cols; j++) {
                    FiniteField sum = FiniteField::fromParts(0, 0);
                    for (int k = 0; k < cols; k++) {
                        sum = (sum + result.at(i, k) * base.at(k, j)).mod();
                    }
                    temp.at(i, j) = sum;
                }
            }
            result = temp;
        }
        
        if (k > 1) {
            printf("Device: Squaring base in power calculation, k=%d\n", k);
            
            // 直接计算矩阵平方
            for (int i = 0; i < rows; i++) {
                for (int j = 0; j < cols; j++) {
                    FiniteField sum = FiniteField::fromParts(0, 0);
                    for (int k = 0; k < cols; k++) {
                        sum = (sum + base.at(i, k) * base.at(k, j)).mod();
                    }
                    temp.at(i, j) = sum;
                }
            }
            base = temp;
        }
        
        k >>= 1;
    }
    
    printf("Device: Completed power calculation\n");
    return result;
}

__device__ FiniteField Matrix::deviceTrace() const {
    FiniteField trace = FiniteField::fromParts(0, 0);
    for (int i = 0; i < this->getRows(); ++i) {
        trace = trace + this->at(i, i);  // 累加对角线元素
    }
    return trace;
}