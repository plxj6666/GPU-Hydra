#include "hip/hip_runtime.h"
#include "matrix.h"
#include "polynomial.h"
#include <hip/hip_runtime.h>
#include <random>
#include <sstream>
#include <cstdio>

#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// CUDA核函数 - 矩阵加法
__global__ void matrixAddKernel(const Matrix* A, const Matrix* B, Matrix* C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < A->getCols() && idy < A->getRows()) {
        C->at(idy, idx) = A->at(idy, idx) + B->at(idy, idx);
    }
}

// CUDA核函数 - 矩阵乘法
__global__ void matrixMultiplyKernel(const Matrix* A, const Matrix* B, Matrix* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < A->getRows() && col < B->getCols()) {
        FiniteField sum = FiniteField::fromParts(0,0);
        
        for (int k = 0; k < A->getCols(); k++) {
            FiniteField a = A->at(row, k);
            FiniteField b = B->at(k, col);
            FiniteField prod = a * b;
            sum = sum + prod;
        }
        
        C->at(row, col) = sum;
    }
}

// 构造函数
__host__ __device__ Matrix::Matrix(int r, int c) : rows(r), cols(c), is_device(false) {
    #ifdef __CUDA_ARCH__
        // 在设备端
        data = (FiniteField*)malloc(rows * cols * sizeof(FiniteField));
        if (data == nullptr) {
            printf("Device memory allocation failed\n");  // 添加调试输出
            rows = 0;
            cols = 0;
            return;
        }
        // 添加内存初始化验证
        for (int i = 0; i < rows * cols; i++) {
            data[i] = FiniteField();
            // 验证初始化是否成功
            if (i == 0) {
                FiniteField test = data[i];
                // 验证加法操作
                FiniteField sum = test + test;
            }
        }
    #else
        // 在主机端
        hipError_t err = hipHostMalloc(&data, rows * cols * sizeof(FiniteField));
        if (err != hipSuccess || data == nullptr) {
            printf("Host memory allocation failed: %s\n", 
                   hipGetErrorString(err));  // 添加错误信息
            throw std::runtime_error("Failed to allocate host memory for matrix");
        }
        // 添加内存初始化验证
        for (int i = 0; i < rows * cols; i++) {
            new (&data[i]) FiniteField();
            // 验证初始化是否成功
            if (i == 0) {
                FiniteField test = data[i];
                // 验证加法操作
                FiniteField sum = test + test;
            }
        }
    #endif
}

// 拷贝构造函数
__host__ __device__ Matrix::Matrix(const Matrix& other) : 
    rows(other.rows), cols(other.cols), is_device(other.is_device) {
    size_t size = rows * cols * sizeof(FiniteField);
    
    #ifdef __CUDA_ARCH__
        data = (FiniteField*)malloc(size);
        if (data != nullptr) {
            memcpy(data, other.data, size);
        }
    #else
        if (is_device) {
            hipMalloc(&data, size);
        } else {
            hipHostMalloc(&data, size);
        }
        if (data != nullptr) {
            if (is_device) {
                hipMemcpy(data, other.data, size, hipMemcpyDeviceToDevice);
            } else {
                memcpy(data, other.data, size);
            }
        }
    #endif
}

// 赋值运算符
__host__ __device__ Matrix& Matrix::operator=(const Matrix& other) {
    if (this != &other) {
        // 先释放原有内存
        if (data != nullptr) {
            #ifdef __CUDA_ARCH__
                free(data);
            #else
                if (is_device) {
                    hipFree(data);
                } else {
                    hipHostFree(data);
                }
            #endif
        }
        
        rows = other.rows;
        cols = other.cols;
        is_device = other.is_device;
        
        size_t size = rows * cols * sizeof(FiniteField);
        
        #ifdef __CUDA_ARCH__
            data = (FiniteField*)malloc(size);
            if (data != nullptr) {
                memcpy(data, other.data, size);
            }
        #else
            if (is_device) {
                hipMalloc(&data, size);
            } else {
                hipHostMalloc(&data, size);
            }
            if (data != nullptr) {
                if (is_device) {
                    hipMemcpy(data, other.data, size, hipMemcpyDeviceToDevice);
                } else {
                    memcpy(data, other.data, size);
                }
            }
        #endif
    }
    return *this;
}

// 在设备上创建矩阵
__host__ Matrix Matrix::createDeviceMatrix(int rows, int cols) {
    Matrix matrix(rows, cols);
    CHECK_CUDA_ERROR(hipMalloc(&matrix.data, rows * cols * sizeof(FiniteField)));
    matrix.is_device = true;
    return matrix;
}

// GPU矩阵加法
__host__ Matrix Matrix::deviceAdd(const Matrix& A, const Matrix& B) {
    if (A.cols != B.cols || A.rows != B.rows) {
        throw std::runtime_error("Matrix dimensions do not match");
    }
    
    if (!A.is_device || !B.is_device) {
        throw std::runtime_error("Input matrices must be on device");
    }
    
    Matrix C = createDeviceMatrix(A.rows, A.cols);
    
    Matrix *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc(&d_A, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, sizeof(Matrix)));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_A, &A, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, &B, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_C, &C, sizeof(Matrix), hipMemcpyHostToDevice));
    
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (A.getCols() + blockSize.x - 1) / blockSize.x,
        (A.getRows() + blockSize.y - 1) / blockSize.y
    );
    
    matrixAddKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return C;
}

// GPU矩阵乘法
__host__ Matrix Matrix::deviceMultiply(const Matrix& A, const Matrix& B) {
    if (A.cols != B.rows) {
        throw std::runtime_error("Matrix dimensions do not match for multiplication");
    }
    
    if (!A.is_device || !B.is_device) {
        throw std::runtime_error("Input matrices must be on device");
    }
    
    Matrix C = createDeviceMatrix(A.rows, B.cols);
    
    Matrix *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc(&d_A, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, sizeof(Matrix)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, sizeof(Matrix)));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_A, &A, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, &B, sizeof(Matrix), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_C, &C, sizeof(Matrix), hipMemcpyHostToDevice));
    
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (B.cols + blockSize.x - 1) / blockSize.x,
        (A.rows + blockSize.y - 1) / blockSize.y
    );
    
    matrixMultiplyKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return C;
}

// 内存拷贝方法
__host__ void Matrix::copyToDevice(Matrix& d_matrix) const {
    size_t size = rows * cols * sizeof(FiniteField);
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.data, data, size, hipMemcpyHostToDevice));
}

__host__ void Matrix::copyFromDevice(const Matrix& d_matrix) {
    if (!d_matrix.is_device) {
        throw std::runtime_error("Source matrix is not a device matrix");
    }
    
    if (d_matrix.rows != rows || d_matrix.cols != cols) {
        throw std::runtime_error("Matrix dimensions do not match for copy");
    }
    
    size_t size = rows * cols * sizeof(FiniteField);
    CHECK_CUDA_ERROR(hipMemcpy(data, d_matrix.data, size, hipMemcpyDeviceToHost));
}

// 随机化矩阵元素
__host__ void Matrix::randomize() {
    std::random_device rd;
    std::mt19937_64 gen(rd());
    std::uniform_int_distribution<uint64_t> dis;
    
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            uint128_t random_value = dis(gen);
            random_value = (random_value << 64) | dis(gen);
            at(i, j) = FiniteField::fromParts(0, random_value);
        }
    }
}

// 转换为字符串
__host__ std::string Matrix::toString() const {
    std::ostringstream oss;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            oss << "M[" << i << "][" << j << "] = ";
            data[i * cols + j].print(oss);
            oss << "\n";
        }
    }
    return oss.str();
}

// 添加一个辅助函数来验证设备内存
__host__ bool Matrix::validateDevicePointer() const {
    hipPointerAttribute_t attrs;
    return hipPointerGetAttributes(&attrs, data) == hipSuccess && 
           attrs.type == hipMemoryTypeDevice;
}

__host__ __device__ bool Matrix::isInvertible() const {
    if (rows != cols) {
        return false;  // 非方阵一定不可逆
    }
    
    // 已经解决列数大于8问题
    // #ifdef __CUDA_ARCH__
    // if (rows > 8) {
    //     // 在设备端，我们不能抛出异常
    //     return false;
    // }
    // #else
    // if (rows > 4) {
    //     throw std::runtime_error("Matrix size larger than 4x4 is not supported");
    // }
    // #endif
    
    FiniteField det = determinant();
    return !(det == FiniteField::fromParts(0,0));
}

__device__ __host__ FiniteField Matrix::determinant() const {
    if (rows != cols) {
        return FiniteField::fromParts(0,0);
    }
    
    // 1x1到4x4矩阵保持原直接计算方式
    if (rows <= 4) {
        return determinantSmall();
    }
    
    // 对于更大的矩阵，使用LU分解
    return determinantLU();
}

__device__ __host__ FiniteField Matrix::determinantSmall() const {
    if (rows == 1) return at(0, 0);
    if (rows == 2) return at(0,0) * at(1,1) - at(0,1) * at(1,0);
    
    if (rows == 3) {
        FiniteField det;
        // 正项
        det = det + (at(0,0) * at(1,1) * at(2,2));
        det = det + (at(0,1) * at(1,2) * at(2,0));
        det = det + (at(0,2) * at(1,0) * at(2,1));
        // 负项
        det = det - (at(0,2) * at(1,1) * at(2,0));
        det = det - (at(0,1) * at(1,0) * at(2,2));
        det = det - (at(0,0) * at(1,2) * at(2,1));
        return det;
    }
    
    if (rows == 4) {
        FiniteField det;
        for (int j = 0; j < 4; j++) {
            FiniteField minor_det;
            if (j == 0) {
                minor_det = (at(1,1) * at(2,2) * at(3,3) + 
                           at(1,2) * at(2,3) * at(3,1) + 
                           at(1,3) * at(2,1) * at(3,2)) -
                          (at(1,3) * at(2,2) * at(3,1) + 
                           at(1,2) * at(2,1) * at(3,3) + 
                           at(1,1) * at(2,3) * at(3,2));
            } else if (j == 1) {
                minor_det = (at(1,0) * at(2,2) * at(3,3) + 
                           at(1,2) * at(2,3) * at(3,0) + 
                           at(1,3) * at(2,0) * at(3,2)) -
                          (at(1,3) * at(2,2) * at(3,0) + 
                           at(1,2) * at(2,0) * at(3,3) + 
                           at(1,0) * at(2,3) * at(3,2));
                minor_det = -minor_det;
            } else if (j == 2) {
                minor_det = (at(1,0) * at(2,1) * at(3,3) + 
                           at(1,1) * at(2,3) * at(3,0) + 
                           at(1,3) * at(2,0) * at(3,1)) -
                          (at(1,3) * at(2,1) * at(3,0) + 
                           at(1,1) * at(2,0) * at(3,3) + 
                           at(1,0) * at(2,3) * at(3,1));
            } else {
                minor_det = (at(1,0) * at(2,1) * at(3,2) + 
                           at(1,1) * at(2,2) * at(3,0) + 
                           at(1,2) * at(2,0) * at(3,1)) -
                          (at(1,2) * at(2,1) * at(3,0) + 
                           at(1,1) * at(2,0) * at(3,2) + 
                           at(1,0) * at(2,2) * at(3,1));
                minor_det = -minor_det;
            }
            det = det + (at(0,j) * minor_det);
        }
        return det;
    }
    
    return FiniteField::fromParts(0,0);
}

__device__ __host__ FiniteField Matrix::determinantLU() const {
    // 使用栈内存代替设备内存
    FiniteField L_data[64];  // 支持最大8x8矩阵
    FiniteField U_data[64];
    
    // 初化L和U
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            if(i == j) {
                L_data[idx] = FiniteField::fromParts(0,1);
            } else {
                L_data[idx] = FiniteField::fromParts(0,0);
            }
            U_data[idx] = FiniteField::fromParts(0,0);
        }
    }
    
    // LU分解
    for(int i = 0; i < rows; i++) {
        // 计算U的第i行
        for(int j = i; j < cols; j++) {
            FiniteField sum = FiniteField::fromParts(0,0);
            for(int k = 0; k < i; k++) {
                sum = sum + L_data[i * cols + k] * U_data[k * cols + j];
            }
            U_data[i * cols + j] = at(i,j) - sum;
        }
        
        // 检查对角线元素是否为0
        if(U_data[i * cols + i].isZero()) {
            return FiniteField::fromParts(0,0); // 矩阵奇异
        }
        
        // 计算L的第i列
        FiniteField u_ii_inv = U_data[i * cols + i].inverse();
        for(int j = i + 1; j < rows; j++) {
            FiniteField sum = FiniteField::fromParts(0,0);
            for(int k = 0; k < i; k++) {
                sum = sum + L_data[j * cols + k] * U_data[k * cols + i];
            }
            L_data[j * cols + i] = (at(j,i) - sum) * u_ii_inv;
        }
    }
    
    // 计算行列式：对角线元素的乘积
    FiniteField det = FiniteField::fromParts(0,1);
    for(int i = 0; i < rows; i++) {
        det = det * U_data[i * cols + i];
    }
    
    return det;
}

__host__ __device__ Matrix::~Matrix() {
    if (!is_device && data != nullptr) {
        delete[] data;
    }
}