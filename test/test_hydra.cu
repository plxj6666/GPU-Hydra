#include <iostream>
#include "Hydra.h"
#include <hip/hip_runtime.h>

int main() {
    // 初始化参数
    uint128_t p_ = uint128_t(0x8000000000000000ULL) << 64 | uint128_t(0x000000000000002dULL);
    FiniteField p = FiniteField::fromParts(0, p_);
    int t = 5;   // 明文长度
    int sec = 128; // 安全级别

    // 主机端初始化 Hydra
    Hydra h_hydra(p, t, sec);

    // 将 Hydra 复制到设备
    Hydra* d_hydra = h_hydra.copyToDevice();

    // 初始化主机端输入数据
    FiniteFieldArray h_state_in(t);
    for (int i = 0; i < t; ++i) {
        h_state_in[i] = FiniteField::fromParts(0, i); // 输入为 0, 1, 2, ...
    }

    // 在设备端分配输出数组
    FiniteFieldArray* d_state_out;
    hipMalloc(&d_state_out, sizeof(FiniteFieldArray));   
    FiniteField* d_elements;
    hipMalloc(&d_elements, sizeof(FiniteField) * t); // 为 elements 分配设备内存
    FiniteFieldArray h_state_out(t); // 在主机端初始化
    h_state_out.setElements(d_elements, false);      // 设置 elements 指针（不拥有内存）

    hipMemcpy(d_state_out, &h_state_out, sizeof(FiniteFieldArray), hipMemcpyHostToDevice);

    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录开始时间
    hipEventRecord(start);

    // 调用加密核函数
    hydraEncrypt<<<1, 1>>>(d_state_out, d_hydra, t);

    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 计算时间差
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time taken for hydraEncrypt: " << milliseconds << " ms" << std::endl;

    // 清理
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}